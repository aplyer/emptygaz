#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_sample.h"


template <typename T, typename X>
inline __host__ __device__ T iDivUp(const T a, const X b) {
  const T c = (T)b;
  const T mod = a - c * (a / c);  // a % c;
  const T div = a / c;
  if (mod != 0)
    return div + 1;
  else
    return div;
}

__global__ void un_kernel(float *a, int N){
  int offset = threadIdx.x + blockIdx.x *  blockDim.x;
  if(offset < N)
    a[offset] = 42;
}

void une_fonction(float *a, int N){
  dim3 grid(iDivUp(N, 512));
  dim3 threads(512);
  un_kernel<<<grid, threads>>> (a, N);
}
